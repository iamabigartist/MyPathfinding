#include "heatmap.cuh"
#define BOUND 40

void gen_map(std::uint32_t *map, std::uint32_t size)
{
  for (std::uint32_t i = 0; i < size; i++)
  {
    if (rand() % 100 > BOUND)
      map[i] = UNREACHABLE;
    else
      map[i] = REACHABLE;
  }
}

void test_prefix_sum()
{
  std::uint32_t s = 768968, res;
  std::uint32_t *arr, *one_for_all;
  hipMalloc(&arr, s * sizeof(std::uint32_t));
  one_for_all = new std::uint32_t[s];
  for (std::uint32_t i = 0; i < s; i++)
    one_for_all[i] = 1;
  hipMemcpy(arr, one_for_all, s * sizeof(std::uint32_t), hipMemcpyHostToDevice);
  prefix_sum(arr, s);
  hipMemcpy(&res, &arr[s - 1], sizeof(std::uint32_t), hipMemcpyDeviceToHost);
  printf("%d\n", res);
  delete[] one_for_all;
  hipFree(arr);
}

void print_heatmap(std::uint32_t *heatmap, std::uint32_t length, std::uint32_t width)
{
  for (std::uint32_t i = 0; i < width; i++)
  {
    for (std::uint32_t j = 0; j < length; j++)
    {
      printf("%3d ", heatmap[i * length + j]);
    }
    printf("\n");
  }
}

void init_heatmap(std::uint32_t *heatmap, std::uint32_t size, std::uint32_t target)
{
  memset(heatmap, 0xFF, size * sizeof(std::uint32_t));
  heatmap[target] = 0;
}

void test_heatmap()
{
  std::uint32_t width = 1 << 5, length = 1 << 5, size = width * length, target = rand() % size;
  printf("Target: %d row %d line\n", target / length, target % length);
  std::uint32_t *d_map, *map, *d_heatmap, *heatmap;
  hipMalloc(&d_map, size * sizeof(std::uint32_t));
  hipMalloc(&d_heatmap, size * sizeof(std::uint32_t));
  map = new std::uint32_t[size];
  heatmap = new std::uint32_t[size];
  gen_map(map, size);
  init_heatmap(heatmap, size, target);
  hipMemcpy(d_map, map, size * sizeof(std::uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_heatmap, heatmap, size * sizeof(std::uint32_t), hipMemcpyHostToDevice);
  heatmap_build(d_map, d_heatmap, width, length, target);
  hipMemcpy(heatmap, d_heatmap, size * sizeof(std::uint32_t), hipMemcpyDeviceToHost);
  print_heatmap(map, length, width);
  printf("\n");
  print_heatmap(heatmap, length, width);
  delete[] map;
  delete[] heatmap;
  hipFree(d_map);
  hipFree(d_heatmap);
}

int main()
{
  //test_prefix_sum();
  test_heatmap();
}